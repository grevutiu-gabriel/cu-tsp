#include "hip/hip_runtime.h"
/*
CUDA TSP solver
Tuomas Rintamäki 2016
tuomas.rintamaki@aalto.fi
*/

/*
License for the helper code for reading the TSPLIB files:

Copyright (c) 2014, Texas State University. All rights reserved.

Redistribution and use in source and binary forms, with or without modification,
are permitted for academic, research, experimental, or personal use provided
that the following conditions are met:

   * Redistributions of source code must retain the above copyright notice,
     this list of conditions and the following disclaimer.
   * Redistributions in binary form must reproduce the above copyright notice,
     this list of conditions and the following disclaimer in the documentation
     and/or other materials provided with the distribution.
   * Neither the name of Texas State University nor the names of its
     contributors may be used to endorse or promote products derived from this
     software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Author: Martin Burtscher
*/

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <limits.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define dist(a, b) __float2int_rn(sqrtf((px[a] - px[b]) * (px[a] - px[b]) + (py[a] - py[b]) * (py[a] - py[b])))
#define swap(a, b) {int tmp = a;  a = b;  b = tmp;}

static __device__ volatile int best_d;
static __device__ volatile int sol_d;

__global__ void Init()
{
  sol_d = 0;
  best_d = INT_MAX;
}

__global__ void TwoOpt(int cities, float *posx, float *posy, float *px, float *py, int *tour, int *len)
{
  int a,b,c,d, Dab;
  int i,j,ii,jj,mini,minj,from,to,cost,offset;
  int minchange, change;

  offset = blockIdx.x*cities;

  // copy the city coordinates and set the initial tour for each block
  for (i = 0; i < cities; i++) {
    px[offset+i] = posx[i];
    py[offset+i] = posy[i];
    tour[offset+i] = i;
  }

  // do serial permutation of the city coordinates so that initial tour is randomized
  hiprandState rndstate;
  hiprand_init(blockIdx.x, 0, 0, &rndstate);
  for (i = 0; i < cities; i++) {
    j = hiprand(&rndstate) % (cities);
    swap(tour[offset+i], tour[offset+j]);
  }
  
  // search for 2-opt moves
  do {
      minchange = 0;
      i = 0;
      b = tour[offset+cities-1];

      while (i < cities-3) {
          a = b;
          i = i+1;
          b = tour[offset+i];
          Dab = dist(a,b);
          j = i+1;
          d = tour[offset+j];

          while (j < cities-1) {
              c = d;
              j = j+1;
              d = tour[offset+j];

              change = dist(a,c) - dist(c,d) + dist(b,d) - Dab;
              if (change < minchange) {
                  //printf("change: %d \n", change);
                  //printf("minchange: %d \n", minchange);
                  minchange = change;
                  mini = i;
                  minj = j;
              }
          }
      }

      // apply the best move
      if (minchange < 0) {
        i = mini;
        j = minj-1;

        while (i < j) {
          swap(tour[offset+j], tour[offset+i]);
          i++;
          j--;
        }
      }
  } while (minchange < 0);

  // we have a local minimum so compute the cost of the tour
  cost = 0;
  for (i = 0;i < cities - 1;i++) {
    from = tour[offset+i];
    to = tour[offset+i+1];
    cost += dist(from,to);
  }

  // check if the current local minimum is the best solution so far and save it if necessary
  atomicMin((int *)&best_d, cost);
  if (best_d == cost) {
    sol_d = blockIdx.x;
  }
}


/******************************************************************************/
/*** helper code **************************************************************/
/******************************************************************************/

static void CudaTest(char *msg)
{
  hipError_t e;

  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "%s: %d\n", msg, e);
    fprintf(stderr, "%s\n", hipGetErrorString(e));
    exit(-1);
  }
}

#define mallocOnGPU(addr, size) if (hipSuccess != hipMalloc((void **)&addr, size)) fprintf(stderr, "could not allocate GPU memory\n");  CudaTest("couldn't allocate GPU memory");
#define copyToGPU(to, from, size) if (hipSuccess != hipMemcpy(to, from, size, hipMemcpyHostToDevice)) fprintf(stderr, "copying of data to device failed\n");  CudaTest("data copy to device failed");
#define copyFromGPU(to, from, size) if (hipSuccess != hipMemcpy(to, from, size, hipMemcpyDeviceToHost)) fprintf(stderr, "copying of data from device failed\n");  CudaTest("data copy from device failed");
#define copyFromGPUSymbol(to, from, size) if (hipSuccess != hipMemcpyFromSymbol(to, HIP_SYMBOL(from), size)) fprintf(stderr, "copying of symbol from device failed\n");  CudaTest("symbol copy from device failed");
#define copyToGPUSymbol(to, from, size) if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(to), from, size)) fprintf(stderr, "copying of symbol to device failed\n");  CudaTest("symbol copy to device failed");

/******************************************************************************/
/*** read TSPLIB input ********************************************************/
/******************************************************************************/

static int readInput(char *fname, float **posx_d, float **posy_d)  // ATT and CEIL_2D edge weight types are not supported
{
  int ch, cnt, in1, cities, i, j;
  float in2, in3;
  FILE *f;
  float *posx, *posy;
  char str[256];  // potential for buffer overrun

  f = fopen(fname, "rt");
  if (f == NULL) {fprintf(stderr, "could not open file %s\n", fname);  exit(-1);}

  ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
  ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
  ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);

  ch = getc(f);  while ((ch != EOF) && (ch != ':')) ch = getc(f);
  fscanf(f, "%s\n", str);
  cities = atoi(str);
  if (cities <= 2) {fprintf(stderr, "only %d cities\n", cities);  exit(-1);}

  posx = (float *)malloc(sizeof(float) * cities);  if (posx == NULL) {fprintf(stderr, "cannot allocate posx\n");  exit(-1);}
  posy = (float *)malloc(sizeof(float) * cities);  if (posy == NULL) {fprintf(stderr, "cannot allocate posy\n");  exit(-1);}

  ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
  fscanf(f, "%s\n", str);
  if (strcmp(str, "NODE_COORD_SECTION") != 0) {fprintf(stderr, "wrong file format\n");  exit(-1);}

  cnt = 0;
  while (fscanf(f, "%d %f %f\n", &in1, &in2, &in3)) {
    posx[cnt] = in2;
    posy[cnt] = in3;
    cnt++;
    if (cnt > cities) {fprintf(stderr, "input too long\n");  exit(-1);}
    if (cnt != in1) {fprintf(stderr, "input line mismatch: expected %d instead of %d\n", cnt, in1);  exit(-1);}
  }
  if (cnt != cities) {fprintf(stderr, "read %d instead of %d cities\n", cnt, cities);  exit(-1);}

  fscanf(f, "%s", str);
  if (strcmp(str, "EOF") != 0) {fprintf(stderr, "didn't see 'EOF' at end of file\n");  exit(-1);}

  mallocOnGPU(*posx_d, sizeof(float) * cities);
  mallocOnGPU(*posy_d, sizeof(float) * cities);
  copyToGPU(*posx_d, posx, sizeof(float) * cities);
  copyToGPU(*posy_d, posy, sizeof(float) * cities);

  fclose(f);
  free(posx);
  free(posy);

  return cities;
}

int main(int argc, char *argv[])
{
  printf("2-opt TSP CUDA GPU code v0.001 \n");

  int cities, restarts, climbs, best, sol;
  int *tour;
  int *tour_d, *len_d;
  float *posx_d, *posy_d, *px_d, *py_d;
  double runtime;
  struct timeval starttime, endtime;

  if (argc != 3) {fprintf(stderr, "\narguments: input_file restart_count\n"); exit(-1);}
  cities = readInput(argv[1], &posx_d, &posy_d);
  restarts = atoi(argv[2]);
  if (restarts < 1) {fprintf(stderr, "restart_count is too small: %d\n", restarts); exit(-1);}

  printf("configuration: %d cities, %d restarts, %s input\n", cities, restarts, argv[1]);

  hipFuncSetCacheConfig(reinterpret_cast<const void*>(TwoOpt), hipFuncCachePreferEqual);

  // allocate memory for saving blockwise x and y positions on the device as well as the tour orders
  mallocOnGPU(px_d, restarts*cities*sizeof(float));
  mallocOnGPU(py_d, restarts*cities*sizeof(float));
  mallocOnGPU(tour_d, restarts*cities*sizeof(int));
  // also, allocate memory for saving the blockwise tour lengths and the final solution
  mallocOnGPU(len_d, restarts*sizeof(int));
  mallocOnGPU(sol_d, cities*sizeof(int));

  gettimeofday(&starttime, NULL);
  Init<<<1, 1>>>();
  TwoOpt<<<restarts, 1>>>(cities, posx_d, posy_d, px_d, py_d, tour_d, len_d);
  CudaTest("kernel launch failed");
  gettimeofday(&endtime, NULL);
  runtime = endtime.tv_sec + endtime.tv_usec / 1000000.0 - starttime.tv_sec - starttime.tv_usec / 1000000.0;

  // read results
  copyFromGPUSymbol(&best, best_d, sizeof(int));
  copyFromGPUSymbol(&sol, sol_d, sizeof(int));
  tour = (int *)malloc(sizeof(int)*cities);  if (tour == NULL) {fprintf(stderr, "cannot allocate tour\n");  exit(-1);}
  copyFromGPU(tour, &tour_d[sol*cities], sizeof(int)*cities);

  // output results
  printf("best instance = %d \n", sol);
  printf("best found tour length = %d\n", best);
  for (int i = 0; i < cities; i++) {
    printf("node %d \n", tour[i]);
  }

  fflush(stdout);

  hipFree(posx_d);
  hipFree(posy_d);
  return 0;
}
